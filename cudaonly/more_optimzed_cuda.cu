#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>

// CUDA device function for heavy integrand
__host__ __device__ double f(double x) {
    return x * x + sin(x) + log(1.0 + x);
}

// CUDA kernel for trapezoid integration
__global__ void trapezoid_kernel(double a, double h, int n, double* d_partial) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= 1 && idx < n) {
        double x = a + idx * h;
        d_partial[idx] = f(x);
    }
}

// Host version of the function (for serial baseline)
double f_host(double x) {
    return x * x + std::sin(x) + std::log(1.0 + x);
}

__host__ int main() {
    const int n = 100000000;
    const double a = 0.0, b = 1.0;
    const double h = (b - a) / n;

    std::vector<int> block_sizes = {128, 256, 512, 1024};

    std::cout << "Mode\tBlockSize\tTime (s)\tIntegral\n";
    std::cout << "----\t---------\t--------\t--------\n";

    // --- Serial Baseline ---
    {
        double sum = 0.0;
        double t0 = clock() / (double)CLOCKS_PER_SEC;

        for (int i = 1; i < n; ++i) {
            double x = a + i * h;
            sum += f_host(x);
        }
        sum += (f_host(a) + f_host(b)) / 2.0;
        double result = h * sum;

        double t1 = clock() / (double)CLOCKS_PER_SEC;
        std::cout << "Serial\t-\t\t" << (t1 - t0) << "\t" << result << "\n";
    }

    // Allocate GPU memory
    double* d_partial;
    hipMalloc(&d_partial, n * sizeof(double));

    std::vector<double> h_partial(n, 0.0);

    for (int blockSize : block_sizes) {
        int gridSize = (n + blockSize - 1) / blockSize;

        hipMemset(d_partial, 0, n * sizeof(double));

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        trapezoid_kernel<<<gridSize, blockSize>>>(a, h, n, d_partial);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float ms = 0;
        hipEventElapsedTime(&ms, start, stop);  // milliseconds

        // Copy results to CPU
        hipMemcpy(h_partial.data(), d_partial, n * sizeof(double), hipMemcpyDeviceToHost);

        // Sum on CPU
        double sum = 0.0;
        for (int i = 1; i < n; ++i)
            sum += h_partial[i];

        sum += (f_host(a) + f_host(b)) / 2.0;
        double result = h * sum;

        std::cout << "CUDA\t" << blockSize << "\t\t" << ms / 1000.0 << "\t" << result << "\n";

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    hipFree(d_partial);
    return 0;
}
