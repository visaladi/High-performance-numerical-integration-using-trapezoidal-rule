#include <iostream>
#include <hip/hip_runtime.h>
#include <omp.h>

__host__ __device__ double f(double x) {
    return x * x;  // function to integrate
}

__global__ void integrate_kernel(double a, double h, int n, double* partial_sums) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= 1 && i < n) {
        double x = a + i * h;
        partial_sums[i] = f(x);
    }
}

int main() {
    const int n = 1000000;
    const double a = 0.0, b = 1.0;
    const double h = (b - a) / n;

    double* d_partial_sums;
    double* h_partial_sums = new double[n];

    // Allocate memory on GPU
    hipMalloc(&d_partial_sums, n * sizeof(double));
    hipMemset(d_partial_sums, 0, n * sizeof(double));

    // Launch CUDA kernel
    int threadsPerBlock = 256;
    int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;
    integrate_kernel<<<blocks, threadsPerBlock>>>(a, h, n, d_partial_sums);
    hipDeviceSynchronize();

    // Copy results back to CPU
    hipMemcpy(h_partial_sums, d_partial_sums, n * sizeof(double), hipMemcpyDeviceToHost);

    // Final summation on CPU using OpenMP
    double sum = 0.0;

    #pragma omp parallel for reduction(+:sum)
    for (int i = 1; i < n; i++) {
        sum += h_partial_sums[i];
    }

    sum += (f(a) + f(b)) / 2.0;
    double result = h * sum;

    std::cout << "Integral estimate (CUDA + OpenMP) = " << result << std::endl;

    // Cleanup
    delete[] h_partial_sums;
    hipFree(d_partial_sums);

    return 0;
}
